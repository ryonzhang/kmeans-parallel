#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <unistd.h>
#include <math.h>
#include <float.h>
#include <string.h>


#define MAX_ITR 10          /* Maximum number of iterations */ 

typedef struct {            /* 2D vector type */
    double x;
    double y;
} Vector;


__device__ int itr = 0;     /* Iteration count */
int     h_numcenters = 4;   /* Host-side center count */
int     h_numpoints;        /* Host-side point count */
double  h_threshold = 0.05; /* Host-side threshold */
Vector* h_centers;          /* Host-side centers */
Vector* h_tmpcenters;       /* Host-side temporary centers */
Vector* h_points;           /* Host-side points */
int*    h_counts;           /* Host-side cluster counts */
int     h_converged;        /* Host-side convergence boolean */
Vector* d_centers;          /* Device-side centers */
Vector* d_tmpcenters;       /* Device-side temporary centers */
Vector* d_points;           /* Device-side points */
int*    d_counts;           /* Device-side cluster counts */
int*    d_converged;        /* Device-side convergence boolean */

/*
 * Return a random point
 */
__host__ Vector random_point()
{
    return h_points[rand() % h_numpoints];
}

/*
 * Return a point at (0,0)
 */
__host__ Vector zero_point()
{
    Vector point;
    point.x = 0;
    point.y = 0;
    
    return point;
}

/*
 * Copy the points to the GPU
 */
__host__ void init_points()
{
    hipMalloc((void **) &d_points, sizeof(Vector) * h_numpoints);
    hipMemcpy(d_points, h_points, sizeof(Vector) * h_numpoints,
               hipMemcpyHostToDevice);
}

/*
 * Copy the initial centers to the GPU
 */
__host__ void init_centers()
{
    int i;
    for (i = 0; i < h_numcenters; i++) {
        h_centers[i] = random_point();
        h_tmpcenters[i] = zero_point();
        h_counts[i] = 0;
    }

    /* Copy the centers to the GPU */
    hipMalloc((void **) &d_centers, sizeof(Vector) * h_numcenters);
    hipMalloc((void **) &d_tmpcenters, sizeof(Vector) * h_numcenters);
    hipMalloc((void **) &d_counts, sizeof(int) * h_numcenters);
    hipMemcpy(d_centers, h_centers, sizeof(Vector) * h_numcenters,
               hipMemcpyHostToDevice);
    hipMemcpy(d_tmpcenters, h_tmpcenters, sizeof(Vector) * h_numcenters,
               hipMemcpyHostToDevice);
    hipMemcpy(d_counts, h_counts, sizeof(int) * h_numcenters,
               hipMemcpyHostToDevice);

    /* Initialize the device-side convergence boolean */
    hipMalloc((void **) &d_converged, sizeof(int));
}

/*
 * Read data points from the input file
 */
__host__ void init_dev(char *inputname)
{
    /* Open the input file */
    if (inputname == NULL) {
        fprintf(stderr, "Must provide an input filename\n");
        free(inputname);
        exit(EXIT_FAILURE);
    }
    FILE *inputfile = fopen(inputname, "r");
    if (inputfile == NULL) {
        fprintf(stderr, "Invalid filename\n");
        free(inputname);
        exit(EXIT_FAILURE);
    }

    /* Read the line count */
    char *line = NULL;
    size_t len = 0;
    ssize_t read = getline(&line, &len, inputfile);
    h_numpoints = atoi(line);

    /* Read each data point in */
    h_points = (Vector *) malloc(sizeof(Vector) * h_numpoints);
    while ((read = getline(&line, &len, inputfile)) != -1) {
        char *saveptr;
        char *token;
        token = strtok_r(line, " ", &saveptr);
        int i = atoi(token) - 1;
        
        token = strtok_r(NULL, " ", &saveptr);
        double x = atof(token);

        token = strtok_r(NULL, " ", &saveptr);
        double y = atof(token);

        h_points[i].x = x;
        h_points[i].y = y;
    }
    h_centers = (Vector *) malloc(sizeof(Vector) * h_numcenters);
    h_tmpcenters = (Vector *) malloc(sizeof(Vector) * h_numcenters);
    h_counts = (int *) malloc(sizeof(Vector) * h_numcenters);
    
    /* Initialize the data structures on the GPU */
    init_points();
    init_centers();

    free(line);
    free(inputname);
    free(h_points);
    free(h_centers);
    free(h_tmpcenters);
    free(h_counts);
    fclose(inputfile);
}

/*
 * Reset the temporary centers and counts
 */
__device__ void reset_tmpcenters(Vector *tmpcenters,
                                 int *counts,
                                 int numcenters)
{
    int i;
    for (i = 0; i < numcenters; i++) {
        tmpcenters[i].x = 0;
        tmpcenters[i].y = 0;
        counts[i] = 0;
    }
}

/*
 * Free the device resources
 */
__host__ void free_dev()
{
    hipFree(d_centers);
    hipFree(d_tmpcenters);
    hipFree(d_points);
}

/*
 * Find the nearest center for each point
 */
__device__ int find_nearest_center(Vector *point,
                                   Vector *centers,
                                   Vector *tmpcenters,
                                   int *counts,
                                   int numcenters)
{
    double distance = DBL_MAX;
    int cluster_idx = 0;
    int i;
    for (i = 0; i < numcenters; i++) {
        Vector center = centers[i];
        double d = sqrt(pow(center.x - point->x, 2.0)
                               + pow(center.y - point->y, 2.0));
        if (d < distance) {
            distance = d;
            cluster_idx = i;
        } 
    }
    tmpcenters[cluster_idx].x += point->x;
    tmpcenters[cluster_idx].y += point->y;
    counts[cluster_idx]++;

    return cluster_idx;
}

/*
 * Average each cluster and update their centers
 */
__device__ void average_each_cluster(Vector *tmpcenters,
                                     int *counts,
                                     int numcenters,
                                     Vector *points,
                                     int numpoints)
{
    /* Average each cluster and update their centers */
    int i;
    for (i = 0; i < numcenters; i++) {
        if (counts[i] != 0) {
            double x_avg = tmpcenters[i].x / counts[i];
            double y_avg = tmpcenters[i].y / counts[i];
            tmpcenters[i].x = x_avg;
            tmpcenters[i].y = y_avg;
        }
    }
}

/*
 * Check if the centers have changed
 */
__device__ int centers_changed(Vector *centers,
                               Vector *tmpcenters,
                               int numcenters,
                               int threshold)
{
    int changed = 0;
    int i;
    for (i = 0; i < numcenters; i++) {
        double x_diff = fabs(tmpcenters[i].x - centers[i].x);
        double y_diff = fabs(tmpcenters[i].y - centers[i].y);
        if (x_diff > threshold || y_diff > threshold)
            changed = 1;

        centers[i].x = tmpcenters[i].x;
        centers[i].y = tmpcenters[i].y;
    }
    
    return changed;
}

/*
 * Print the results
 */
__device__ void print_results(Vector *centers,
                              int numcenters)
{
    printf("Converged in %d iterations (max=%d)\n", itr, MAX_ITR);

    int i;
    for (i = 0; i < numcenters; i++)
        printf("Cluster %d center: x=%f, y=%f\n", i, centers[i].x, centers[i].y);
}

/*
 * Compute k-means on the device
 */
__global__ void kmeans_kernel(Vector *points,
			      Vector *centers,
			      Vector *tmpcenters,
			      int *counts,
			      int numcenters,
			      int numpoints,
			      int threshold,
			      int *converged)
{
    /* Re-cluster the points, compute the averages,
     * and check for convergence */
    reset_tmpcenters(tmpcenters, counts, numcenters);
    int i;
    for (i = 0; i < numpoints; i++)
	find_nearest_center(&points[i], centers, tmpcenters, counts, numcenters);
    average_each_cluster(tmpcenters, counts, numcenters, points, numpoints);

    itr++;
    *converged = itr >= MAX_ITR || !centers_changed(centers, tmpcenters, numcenters, threshold);
    if (*converged)
	print_results(centers, numcenters);
}

/*
 * Host-side wrapper for kmeans_kernel
 */
__host__ void kmeans(char *inputname)
{
    init_dev(inputname);
    do {
	kmeans_kernel<<<1, 1>>>(d_points, d_centers, d_tmpcenters, d_counts,
				h_numcenters, h_numpoints, h_threshold,
				d_converged);
	hipMemcpy(&h_converged, d_converged, sizeof(int), hipMemcpyDeviceToHost);
    } while(!h_converged);

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess) {
        fprintf(stderr, "Kernel launch failed with error \"%s\". \n",
                hipGetErrorString(cudaerr));
    }
    free_dev();
}

int main (int argc,
          char *const *argv)
{
    char* inputname;   
    size_t len;
    int opt;
    while ((opt = getopt(argc, argv, "k:t:i:")) != -1) {
        switch (opt) {
        case 'k':
            h_numcenters = atoi(optarg);
            break;
        case 't':
            h_threshold = atof(optarg);
            break;
        case 'i':
            len = strlen(optarg);
            inputname = (char*) malloc(len + 1);
            strcpy(inputname, optarg);
            break;
        default:
            fprintf(stderr, "Usage: %s [-k clusters] [-t threshold]"
                            " [-i inputfile]\n", argv[0]);
            exit(EXIT_FAILURE);
        }
    }
    if (inputname == NULL) {
        fprintf(stderr, "Must provide a valid input filename\n");
        exit(EXIT_FAILURE);
    }
    
    kmeans(inputname);
    return 0;
}