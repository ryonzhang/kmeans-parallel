#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <unistd.h>
#include <math.h>
#include <float.h>
#include <string.h>


typedef struct {    /* A 2D vector */
    double x;
    double y;
    int cluster;
} Vector;


int     _k = 4;            /* Number of clusters */
double  _threshold = 0.05; /* Threshold for convergence */
char*   _inputname;        /* Input filename to read from */
Vector* _centers;          /* Global array of centers */
Vector* _points;           /* Global array of 2D data points */
int     _numpoints;        /* Number of 2D data points */


/*
 * Return a random center to be associated
 * with a cluster
 */
__device__ Vector random_center(int cluster) {
    /* Vector *point = &_points[rand() % _numpoints];
       point->cluster = cluster; */

    return _points[rand() % _numpoints];
}

/*
 * Return a center at (0,0) to be associated
 * with a cluster
 */
__device__ Vector zero_center(int cluster) {
    Vector point;
    point.x = 0;
    point.y = 0;
    point.cluster = cluster;

    return point;
}

/*
 * Create the initial, random centers
 */
__device__ void init_centers(Vector *tmpcenters, Vector *centers) { 
    int i;
    for (i = 0; i < _k; i++) {
	centers[i] = zero_center(i);
	tmpcenters[i] = random_center(i);
    }
}

/*
 * Find the nearest center for each point
 */
__device__ void find_nearest_center(Vector *centers, Vector *point) {
    double distance = DBL_MAX;
    int cluster_idx = 0;
    int i;
    for (i = 0; i < _k; i++) {
	Vector center = centers[i];
	double d = sqrt(pow(center.x - point->x, 2.0)
			       + pow(center.y - point->y, 2.0));
	if (d < distance) {
	    distance = d;
	    cluster_idx = i;
	} 
    }

    point->cluster = cluster_idx;
}

/*
 * Average each cluster and update their centers
 */
__device__ void average_each_cluster(Vector *centers) {
    /* Initialize the arrays */
    double x_sums[_k];
    double y_sums[_k];
    int counts[_k];
    int i;
    for (i = 0; i < _k; i++) {
	x_sums[i] = 0;
	y_sums[i] = 0;
	counts[i] = 0;
    }

    /* Sum up and count each cluster */
    for (i = 0; i < _numpoints; i++) {
	Vector point = _points[i];
	x_sums[point.cluster] += point.x;
	y_sums[point.cluster] += point.y;
	counts[point.cluster] += 1;
    }

    /* Average each cluster and update their centers */
    for (i = 0; i < _k; i++) {
	if (counts[i] != 0) {
	    double x_avg = x_sums[i] / counts[i];
	    double y_avg = y_sums[i] / counts[i];
	    centers[i].x = x_avg;
	    centers[i].y = y_avg;
	} else {
	    centers[i].x = 0;
	    centers[i].y = 0;
	}
    }
}

/*
 * Check if the centers have changed
 */
__device__ int centers_changed(Vector *tmpcenters, Vector *centers) {
    int changed = 0;
    int i;
    for (i = 0; i < _k; i++) {
	double x_diff = fabs(tmpcenters[i].x - centers[i].x);
	double y_diff = fabs(tmpcenters[i].y - centers[i].y);
	if (x_diff > _threshold || y_diff > _threshold) {
	    changed = 1;
	}

	centers[i].x = tmpcenters[i].x;
	centers[i].y = tmpcenters[i].y;
    }

    return changed;
}

/*
 * Compute k-means on the GPU and print out the centers
 */
__global__ void kmeans(Vector *centers, int k, Vector *points, int numpoints) {
    Vector *tmpcenters;
    hipMalloc((Vector **) &tmpcenters, sizeof(Vector) * k);
    init_centers(tmpcenters, centers);

    /* While the centers have moved, re-cluster 
	the points and compute the averages */
    int max_itr = 10;
    int itr = 0;
    while (centers_changed(tmpcenters, centers) && itr < max_itr) {
	int i;
	for (i = 0; i < numpoints; i++) {
	    find_nearest_center(tmpcenters, &points[i]);
	}

	average_each_cluster(tmpcenters);
	itr++;
    }
    printf("Converged in %d iterations (max=%d)\n", itr, max_itr);
    
    /* Print the center of each cluster */
    int j;
    for (j = 0; j < k; j++) {
	printf("Cluster %d center: x=%f, y=%f\n",
	       j, centers[j].x, centers[j].y);
    }

    hipFree(tmpcenters);
}

/*
 * Read data points from the input file
 */
__host__ void read_inputfile(char *inputname) {
    _centers = (Vector *) malloc(sizeof(Vector) * _k);

    /* Open the input file */
    if (_inputname == NULL) {
	fprintf(stderr, "Must provide an input filename\n");
	free(_inputname);
	free(_centers);
	exit(EXIT_FAILURE);
    }
    
    FILE *inputfile = fopen(_inputname, "r");
    if (inputfile == NULL) {
	fprintf(stderr, "Invalid filename\n");
	free(_inputname);
	free(_centers);
	exit(EXIT_FAILURE);
    }

    /* Read the line count */
    char *line = NULL;
    size_t len = 0;
    ssize_t read = getline(&line, &len, inputfile);
    _numpoints = atoi(line);
    _points = (Vector *) malloc(sizeof(Vector) * _numpoints);

    /* Read each data point in */
    while ((read = getline(&line, &len, inputfile)) != -1) {
	char *saveptr;
	char *token;
	token = strtok_r(line, " ", &saveptr);
	int i = atoi(token) - 1;
	
	token = strtok_r(NULL, " ", &saveptr);
	double x = atof(token);

	token = strtok_r(NULL, " ", &saveptr);
	double y = atof(token);

	_points[i].x = x;
	_points[i].y = y;
	_points[i].cluster = 0;
    }
    
    free(line);
    fclose(inputfile);
}

__host__ int main (int argc, char *const *argv) {
    size_t len;
    int opt;
    while ((opt = getopt(argc, argv, "k:t:i:")) != -1) {
	switch (opt) {
	case 'k':
	    _k = atoi(optarg);
	    break;
	case 't':
	    _threshold = atof(optarg);
	    break;
	case 'i':
	    len = strlen(optarg);
	    _inputname = (char*) malloc(len + 1);
	    strcpy(_inputname, optarg);
	    break;
	default:
	    fprintf(stderr, "Usage: %s [-k clusters] [-t threshold]"
                            " [-i inputfile]\n", argv[0]);
	    exit(EXIT_FAILURE);
	}
    }

    read_inputfile(_inputname);
    kmeans<<<1, 1>>>(_centers, _k, _points, _numpoints);

    free(_inputname);
    free(_centers);
    free(_points);
    return 0;
}